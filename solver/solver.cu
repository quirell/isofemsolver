#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <conio.h>
#include <ctime>
#include <new>
#include <cmath>
#include <cstring>
#define ERRCHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true, bool wait = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (wait) getch();
		if (abort) exit(code);
	}
}

#define MSIZE 36
#define INPUT_SIZE(N) N*5 - 3*2
#define PARENT(i) (i-1)/2
#define LEFT(i) 2*i + 1
#define RIGHT(i) 2*i + 2
#define BOTTOM_HEAP_NODES_COUNT(N) (N-2)/3 //size of input must be 2+3n,n>1
#define HEAP_SIZE(N) 2*BOTTOM_HEAP_NODES_COUNT(N)-1 
#define FIRST_LEVEL_SIZE 19
#define ROW_LENGTH 5
#define FIRST_LVL_MAT_SIZE 5
#define XY(x,y) x*6+y
#define THREADS 512
#define BLOCKS(N) (N+THREADS)/THREADS
#define COLUMNS_PER_THREAD 1

const struct Properties
{
	int leftCount;
	int leftSize;
	int heapNodes;
	int bottomNodes;
	int remainingNodes;
	int lastLevelNodes;
	int beforeLastLevelNodes;
	int lastLevelStartIdx;
	int beforeLastLevelStartIdx;
	int rightCount;
	int beforeLastLevelNotBottomNodes;
};

Properties getProperities(int leftCount, int rightCount)
{
	Properties p;
	p.leftCount = leftCount;
	p.leftSize = leftCount * 5;
	p.heapNodes = HEAP_SIZE(leftCount);
	p.bottomNodes = BOTTOM_HEAP_NODES_COUNT(leftCount);
	p.remainingNodes = p.heapNodes - p.bottomNodes;
	p.beforeLastLevelStartIdx = (int)pow(2, (int)log2(p.remainingNodes)) - 1;
	p.beforeLastLevelNotBottomNodes = p.remainingNodes - p.beforeLastLevelStartIdx;
	p.beforeLastLevelNodes = pow(2, (int)log2(p.bottomNodes - 1)) - p.beforeLastLevelNotBottomNodes;// -1 is in case bottomNodes is power of two, then beforeLastLevelNodes should obviously be 0
	p.lastLevelNodes = p.bottomNodes - p.beforeLastLevelNodes;
	p.lastLevelStartIdx = p.heapNodes - p.lastLevelNodes;
	p.beforeLastLevelStartIdx = p.remainingNodes - p.lastLevelNodes; //account for idx value, undefined when beforeLastLevelNodes is = 0
	p.rightCount = rightCount;
	return p;
}


__constant__ Properties dProps;

struct Node
{
	float m[MSIZE];
	float* x[6];
};

__device__ __host__ void printNode(Node node);


__device__ __inline__ int rXY(int x, int y)
{
	return x * dProps.rightCount + y;
}

__global__ void backwardSubstitutionRight(Node* nodes, int startIdx, int nodesCount, int start, int elim)
{
	int idx = (blockIdx.x * blockDim.x + threadIdx.x) / (dProps.rightCount / COLUMNS_PER_THREAD);
	if (idx >= nodesCount)
		return;
	int colStart = ((blockIdx.x * blockDim.x + threadIdx.x) % (dProps.rightCount / COLUMNS_PER_THREAD))*COLUMNS_PER_THREAD;
//	printf("%d %d\n", idx, colStart);
	int nodeIdx = startIdx + idx;
	float* m = nodes[nodeIdx].m;
	float** x = nodes[nodeIdx].x;
	for (int rcol = colStart; rcol < colStart + COLUMNS_PER_THREAD; rcol++)
	{
		for (int row = elim; row >= 0; row--)//max elim == 4,5th is already done after elimination
		{
			for (int col = row + 1; col < 6; col++)
			{
				x[row][rcol] -= m[XY(row, col)] * x[col][rcol];
			}
		}
	}
}

__global__ void forwardEliminationLeft(Node* nodes, int startIdx, int nodesCount, int start, int elim)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= nodesCount)
		return;
	int nodeIdx = startIdx + idx;
	float* m = nodes[nodeIdx].m;
	for (int row = start; row < elim; row++)
	{
		for (int col = row + 1; col < 6; col++) //from element after diagonal
		{
			m[XY(row, col)] /= m[XY(row, row)];
		}
		for (int rowBelow = row + 1; rowBelow < 6; rowBelow++)
		{
			for (int col = row + 1; col < 6; col++)
			{
				m[XY(rowBelow, col)] -= m[XY(rowBelow, row)] * m[XY(row, col)];
			}
		}
	}
}


__global__ void forwardEliminationRight(Node* nodes, int startIdx, int nodesCount, int rowStart, int elim)
{
	int idx = (blockIdx.x * blockDim.x + threadIdx.x) / (dProps.rightCount / COLUMNS_PER_THREAD);
	if (idx >= nodesCount)
		return;
	int colStart = ((blockIdx.x * blockDim.x + threadIdx.x) % (dProps.rightCount / COLUMNS_PER_THREAD))*COLUMNS_PER_THREAD;
	int nodeIdx = startIdx + idx;
	float* m = nodes[nodeIdx].m;
	float** x = nodes[nodeIdx].x;
	for (int row = rowStart; row < elim; row++)
	{
		for (int col = colStart; col < colStart + COLUMNS_PER_THREAD; col++)
		{
			x[row][col] /= m[XY(row, row)];
		}
		for (int rowBelow = row + 1; rowBelow < 6; rowBelow++)
		{
			for (int col = colStart; col < colStart + COLUMNS_PER_THREAD; col++)
			{
				x[rowBelow][col] -= m[XY(rowBelow, row)] * x[row][col];
			}
		}
	}
}

__global__ void assignTestRightSize(Node* node, float* x)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= 1)
		return;
	node->x[0] = x;
	node->x[1] = x+4;
	node->x[2] = x+4*2;
	node->x[3] = x+4*3;
	node->x[4] = x+4*4;
	node->x[5] = x+4*5;
}

void testGaussianElimination()
{
	Properties props = getProperities(1, 4);
	ERRCHECK(hipMemcpyToSymbol(HIP_SYMBOL(dProps), &props, sizeof(Properties)));
	Node node;
	float m[] = {
		1, 1, -2, 1, 3, -1,
		2, -1, 1, 2, 1, -3,
		1, 3, -3, -1, 2, 1,
		5, 2, -1, -1, 2, 1,
		-3, -1, 2, 3, 1, 3,
		4, 3, 1, -6, -3, -2
	};
	memcpy(node.m, m, sizeof(float) * MSIZE);
	Node* dNode;
	printNode(node);
	ERRCHECK(hipMalloc(&dNode, sizeof(Node)));
	ERRCHECK(hipMemcpy(dNode, &node, sizeof(Node), hipMemcpyHostToDevice));
	float x[] = {4,4,4,4,20,20,20,20,-15,-15,-15,-15,-3,-3,-3,-3,16,16,16,16,-27,-27,-27,-27};
	float* dX;
	ERRCHECK(hipMalloc(&dX, sizeof(x)));
	ERRCHECK(hipMemcpy(dX, &x, sizeof(x) , hipMemcpyHostToDevice));
	assignTestRightSize << <1, 1 >> >(dNode, dX);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	forwardEliminationLeft << <1, 1 >> >(dNode, 0, 1, 0, 6);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	forwardEliminationRight << <1, 4/COLUMNS_PER_THREAD >> >(dNode, 0, 1, 0, 6);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	backwardSubstitutionRight << <1, 4/COLUMNS_PER_THREAD >> >(dNode, 0, 1, 0, 4);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	ERRCHECK(hipMemcpy(&node, dNode, sizeof(Node), hipMemcpyDeviceToHost));
	printNode(node);
	ERRCHECK(hipMemcpy(x, dX, sizeof(x), hipMemcpyDeviceToHost));
	float c[] = {1,-2,3,4,2,-1};
	for (int i = 0; i < 6; i++)
	{
		printf("%.1f ", c[i]);
	}
	printf("\n");
	for (int i = 0; i < props.rightCount; i++)
	{
		for (int j= 0; j < 6;j++)
			printf("%.1f ", x[j*props.rightCount+i]);
		printf("\n");
	}
}


__global__ void mergeLeftChild(Node* nodes, int startIdx, int nodesCount)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= nodesCount)
		return;
	int nodeIdx = startIdx + idx;
	Node* parent = &nodes[nodeIdx];

	Node* left = &nodes[LEFT(nodeIdx)];

	parent->m[XY(0, 0)] += left->m[XY(4, 4)];
	parent->m[XY(0, 1)] += left->m[XY(4, 5)];
	parent->m[XY(1, 0)] += left->m[XY(5, 4)];
	parent->m[XY(1, 1)] += left->m[XY(5, 5)];

	parent->m[XY(0, 2)] = left->m[XY(4, 2)];
	parent->m[XY(0, 3)] = left->m[XY(4, 3)];
	parent->m[XY(1, 2)] = left->m[XY(5, 2)];
	parent->m[XY(1, 3)] = left->m[XY(5, 3)];

	parent->m[XY(2, 0)] = left->m[XY(2, 4)];
	parent->m[XY(2, 1)] = left->m[XY(2, 5)];
	parent->m[XY(2, 2)] = left->m[XY(2, 2)];
	parent->m[XY(2, 3)] = left->m[XY(2, 3)];
	parent->m[XY(3, 0)] = left->m[XY(3, 4)];
	parent->m[XY(3, 1)] = left->m[XY(3, 5)];
	parent->m[XY(3, 2)] = left->m[XY(3, 2)];
	parent->m[XY(3, 3)] = left->m[XY(3, 3)];
}

__global__ void mergeRightChild(Node* nodes, int startIdx, int nodesCount)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= nodesCount)
		return;
	int nodeIdx = startIdx + idx;
	Node* parent = &nodes[nodeIdx];
	Node* right = &nodes[RIGHT(nodeIdx)];
	parent->m[XY(0, 0)] += right->m[XY(2, 2)];
	parent->m[XY(0, 1)] += right->m[XY(2, 3)];
	parent->m[XY(1, 0)] += right->m[XY(3, 2)];
	parent->m[XY(1, 1)] += right->m[XY(3, 3)];

	parent->m[XY(0, 4)] = right->m[XY(2, 4)];
	parent->m[XY(0, 5)] = right->m[XY(2, 5)];
	parent->m[XY(1, 4)] = right->m[XY(3, 4)];
	parent->m[XY(1, 5)] = right->m[XY(3, 5)];

	parent->m[XY(4, 0)] = right->m[XY(4, 2)];
	parent->m[XY(4, 1)] = right->m[XY(4, 3)];
	parent->m[XY(4, 4)] = right->m[XY(4, 4)];
	parent->m[XY(4, 5)] = right->m[XY(4, 5)];
	parent->m[XY(5, 0)] = right->m[XY(5, 2)];
	parent->m[XY(5, 1)] = right->m[XY(5, 3)];
	parent->m[XY(5, 4)] = right->m[XY(5, 4)];
	parent->m[XY(5, 5)] = right->m[XY(5, 5)];
}

__global__ void divideLeft(Node* nodes, float* leftSide)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= dProps.bottomNodes)
		return;
	int nodeIdx = (idx < dProps.lastLevelNodes) * (dProps.lastLevelStartIdx + idx) + (idx >= dProps.lastLevelNodes) * (dProps.beforeLastLevelStartIdx + idx);
	//printf("%d %d\n", idx, nodeIdx);
	Node node = nodes[nodeIdx];
	idx *= 5 * 3;
	node.m[XY(2, 3)] = leftSide[idx + 2] / 3;  //maybe swap 2 col with 3 col is also required
	node.m[XY(2, 2)] = leftSide[idx + 3] / 2;
	node.m[XY(2, 1)] = leftSide[idx + 4];

	node.m[XY(3, 3)] = leftSide[idx + 6] / 2;
	node.m[XY(3, 2)] = leftSide[idx + 7] * 2 / 3;
	node.m[XY(3, 1)] = leftSide[idx + 8];
	node.m[XY(3, 4)] = leftSide[idx + 9];

	node.m[XY(1, 3)] = leftSide[idx + 10];
	node.m[XY(1, 2)] = leftSide[idx + 11];
	node.m[XY(1, 1)] = leftSide[idx + 12];
	node.m[XY(1, 4)] = leftSide[idx + 13];
	node.m[XY(1, 5)] = leftSide[idx + 14];

	node.m[XY(4, 2)] = leftSide[idx + 15];
	node.m[XY(4, 1)] = leftSide[idx + 16];
	node.m[XY(4, 4)] = leftSide[idx + 17] * 2 / 3;
	node.m[XY(4, 5)] = leftSide[idx + 18] / 2;

	node.m[XY(5, 1)] = leftSide[idx + 20];
	node.m[XY(5, 4)] = leftSide[idx + 21] / 2;
	node.m[XY(5, 5)] = leftSide[idx + 22] / 3;
	nodes[nodeIdx] = node;
	//	printNode(node);
}

__global__ void divideFirstAndLast(Node* nodes, float* leftSide)
{
	int nodeIdx = dProps.lastLevelStartIdx;
	nodes[dProps.remainingNodes].m[XY(3, 3)] = leftSide[2];
	nodes[dProps.remainingNodes].m[XY(3, 2)] = leftSide[3];

	nodes[dProps.remainingNodes].m[XY(2, 3)] = leftSide[6];
	nodes[dProps.remainingNodes].m[XY(2, 2)] = leftSide[7];

	nodeIdx = (dProps.beforeLastLevelNodes == 0) * (dProps.heapNodes - 1) + (dProps.beforeLastLevelNodes != 0) * (dProps.heapNodes - dProps.lastLevelNodes - 1);
	nodes[nodeIdx].m[XY(4, 4)] = leftSide[dProps.leftSize - 25 + 17];
	nodes[nodeIdx].m[XY(4, 5)] = leftSide[dProps.leftSize - 25 + 18];

	nodes[nodeIdx].m[XY(5, 4)] = leftSide[dProps.leftSize - 25 + 21];
	nodes[nodeIdx].m[XY(5, 5)] = leftSide[dProps.leftSize - 25 + 22];
	//printf("|%d %d|\n", dProps.lastLevelStartIdx, nodeIdx);
}

__global__ void assignRightSideToNodes(Node* nodes, float* rightSide)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	Node node = nodes[idx];
	//	node.x = rightSide + idx*FIRST_LEVEL_SIZE*dRightCols;
}

void leftSideInit(float* leftSide, int size)
{
	for (int i = 0; i < size; i++)
	{
		leftSide[i] = 6;//(i+1)%26;
	}
}

void showMemoryConsumption()
{
	size_t free_byte;
	size_t total_byte;
	ERRCHECK(hipMemGetInfo(&free_byte, &total_byte));
	double free_db = (double)free_byte;
	double total_db = (double)total_byte;
	double used_db = total_db - free_db;

	printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n", used_db / 1024.0 / 1024.0, free_db / 1024.0 / 1024.0, total_db / 1024.0 / 1024.0);
}


void distributeInputAmongNodes(Node* dNodes, float* dLeftSide, Properties props)
{
	divideLeft << <(props.bottomNodes + THREADS) / THREADS, THREADS >> >(dNodes, dLeftSide);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	divideFirstAndLast << <1, 1 >> >(dNodes, dLeftSide);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	//divide right side somehow
}


int eliminateFirstRow(Node* dNodes, Properties props) //5x5 matrices
{
	forwardEliminationLeft << <BLOCKS(props.bottomNodes), THREADS >> >(dNodes, props.lastLevelStartIdx, props.bottomNodes, 1, 1);
	forwardEliminationRight << <BLOCKS(props.bottomNodes), THREADS >> >(dNodes, props.lastLevelStartIdx, props.bottomNodes, 1, 1);
	if (props.beforeLastLevelNodes > 0)
	{
		forwardEliminationLeft << <BLOCKS(props.beforeLastLevelNodes), THREADS >> >(dNodes, props.lastLevelStartIdx, props.bottomNodes, 1, 1); 
		forwardEliminationRight << <BLOCKS(props.beforeLastLevelNodes), THREADS >> >(dNodes, props.lastLevelStartIdx, props.bottomNodes, 1, 1);
	}
	return props.beforeLastLevelNotBottomNodes;
}

void run(Node* dNodes, float* dLeftSide, Properties props, float* leftSize)
{
	distributeInputAmongNodes(dNodes, dLeftSide, props);
	int nodesCount = eliminateFirstRow(dNodes, props);

	for (int start = PARENT(props.lastLevelStartIdx); start > 0; nodesCount = (start + 1) / 2 , start = PARENT(start))//order matters
	{
		mergeLeftChild << <BLOCKS(nodesCount), THREADS >> >(dNodes, start, nodesCount);
		mergeRightChild << <BLOCKS(nodesCount), THREADS >> >(dNodes, start, nodesCount);
		forwardEliminationLeft << <BLOCKS(nodesCount), THREADS >> >(dNodes, start, nodesCount, 0, 2);
		forwardEliminationRight << <BLOCKS(nodesCount*(props.rightCount / COLUMNS_PER_THREAD)), THREADS >> >(dNodes, start, nodesCount, 0, 2);
	}
//	nodesCount = 2;
//	for (int start = 1; start < ; nodesCount = (start + 1) / 2, start = PARENT(start))//order matters
//	{
//
//	}
}


int main()
{
	ERRCHECK(hipSetDevice(0));
	testGaussianElimination();
	getch();
	return 0;
	clock_t start, end;
	//	int leftCount = (3*4+2)*10e5;
	int leftCount = 3 * 3 + 2;
	int rightCount = 1;
	const Properties props = getProperities(leftCount, rightCount);
	ERRCHECK(hipMemcpyToSymbol(HIP_SYMBOL(dProps), &props, sizeof(Properties)));

	float* leftSide = new float[props.leftSize];
	//	float * rightSide = new float[rightSize];
	Node* nodes = new Node[props.heapNodes];
	Node* dNodes = nullptr;
	float* dLeftSide = nullptr;
	leftSideInit(leftSide, props.leftSize);
	ERRCHECK(hipMalloc(&dNodes, sizeof(Node)* props.heapNodes));
	ERRCHECK(hipMemset(dNodes, 0, sizeof(Node)*props.heapNodes));
	ERRCHECK(hipMalloc(&dLeftSide, sizeof(float)*props.leftSize));
	ERRCHECK(hipMemcpy(dLeftSide, leftSide, sizeof(float)*props.leftSize, hipMemcpyHostToDevice));
	showMemoryConsumption();
	start = clock();
	divideLeft << <(props.bottomNodes + 512) / 512, 512 >> >(dNodes, dLeftSide);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	end = clock();
	printf("time %f\n", (float)(end - start) / CLOCKS_PER_SEC);
	divideFirstAndLast << <1, 1 >> >(dNodes, dLeftSide);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	mergeLeftChild << <1, 1 >> >(dNodes, PARENT(props.lastLevelStartIdx), 1);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	mergeRightChild << <1, 1 >> >(dNodes, PARENT(props.lastLevelStartIdx), 1);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	ERRCHECK(hipMemcpy(nodes, dNodes, sizeof(Node) * props.heapNodes, hipMemcpyDeviceToHost));
	printNode(nodes[0]);
	printNode(nodes[1]);
	printNode(nodes[2]);
	printNode(nodes[3]);
	printNode(nodes[4]);
	delete[] nodes;
	hipFree(dNodes);

	ERRCHECK(hipDeviceReset());

	getch();
	return 0;
}


__device__ __host__ void printNode(Node node)
{
	for (int i = 0; i < 6; i++)
		printf("%.1f %.1f %.1f %.1f %.1f %.1f\n", node.m[XY(i, 0)], node.m[XY(i, 1)], node.m[XY(i, 2)], node.m[XY(i, 3)], node.m[XY(i, 4)], node.m[XY(i, 5)]);
	printf("\n");
}
