#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <conio.h>
#include <new>
#include "constants.cuh"
#include "helpers.cuh"
#include "solver.cuh"
#include "test_functions.cuh"
#include <cstring>
#include <ctime>
#include "bitmap_approx.cuh"
#include <string>

__constant__ Properties dProps;

__global__ void backwardSubstitutionRight(Node* nodes, int startIdx, int nodesCount, int end, int elim)
{
	int idx = (blockIdx.x * blockDim.x + threadIdx.x) / (dProps.rightCount / COLUMNS_PER_THREAD);
	if (idx >= nodesCount)
		return;
	int colStart = ((blockIdx.x * blockDim.x + threadIdx.x) % (dProps.rightCount / COLUMNS_PER_THREAD)) * COLUMNS_PER_THREAD;
	//	printf("%d %d\n", idx, colStart);
	int nodeIdx = startIdx + idx;
	number* m = nodes[nodeIdx].m;
	number** x = nodes[nodeIdx].x;
	for (int rcol = colStart; rcol < colStart + COLUMNS_PER_THREAD; rcol++)
	{
		for (int row = elim; row >= end; row--)//max elim == 4,5th is already done after elimination
		{
			for (int col = row + 1; col < 6; col++)
			{
				x[row][rcol] -= m[XY(row, col)] * x[col][rcol];
			}
		}
	}
}

__global__ void forwardEliminationLeft(Node* nodes, int startIdx, int nodesCount, int start, int elim)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= nodesCount)
		return;
	int nodeIdx = startIdx + idx;
	number* m = nodes[nodeIdx].m;
	for (int row = start; row < elim; row++)
	{
		for (int col = row + 1; col < 6; col++) //from element after diagonal
		{
			//printf("%.1f/%.1f = ", m[XY(row,col)], m[XY(row, row)]);
			m[XY(row, col)] /= m[XY(row, row)];
			//			printf("%.2f  ", m[XY(row, col)]);
		}
		//		printf("\n");
		for (int rowBelow = row + 1; rowBelow < 6; rowBelow++)
		{
			for (int col = row + 1; col < 6; col++)
			{
				//printf("%.1f-%.1f*%.1f = ", m[XY(rowBelow, col)], m[XY(rowBelow, row)], m[XY(row, col)]);
				m[XY(rowBelow, col)] -= m[XY(rowBelow, row)] * m[XY(row, col)];
				//				printf("%.2f  ", m[XY(rowBelow, col)]);
			}
			//			printf("\n");
		}
		//		printf("\n\n");
	}
}

__global__ void forwardEliminationRight(Node* nodes, int startIdx, int nodesCount, int rowStart, int elim)
{
	int idx = (blockIdx.x * blockDim.x + threadIdx.x) / (dProps.rightCount / COLUMNS_PER_THREAD);
	if (idx >= nodesCount)
		return;
	int colStart = ((blockIdx.x * blockDim.x + threadIdx.x) % (dProps.rightCount / COLUMNS_PER_THREAD)) * COLUMNS_PER_THREAD;
	int nodeIdx = startIdx + idx;
	number* m = nodes[nodeIdx].m;
	number** x = nodes[nodeIdx].x;
	for (int row = rowStart; row < elim; row++)
	{
		for (int col = colStart; col < colStart + COLUMNS_PER_THREAD; col++)
		{
			x[row][col] /= m[XY(row, row)];
		}
		for (int rowBelow = row + 1; rowBelow < 6; rowBelow++)
		{
			for (int col = colStart; col < colStart + COLUMNS_PER_THREAD; col++)
			{
				x[rowBelow][col] -= m[XY(rowBelow, row)] * x[row][col];
			}
		}
	}
}

__global__ void mergeLeftChild(Node* nodes, int startIdx, int nodesCount)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= nodesCount)
		return;
	int nodeIdx = startIdx + idx;
	Node* parent = &nodes[nodeIdx];

	Node* left = &nodes[LEFT(nodeIdx)];

	parent->m[XY(0, 0)] += left->m[XY(4, 4)];
	parent->m[XY(0, 1)] += left->m[XY(4, 5)];
	parent->m[XY(1, 0)] += left->m[XY(5, 4)];
	parent->m[XY(1, 1)] += left->m[XY(5, 5)];

	parent->m[XY(0, 2)] = left->m[XY(4, 2)];
	parent->m[XY(0, 3)] = left->m[XY(4, 3)];
	parent->m[XY(1, 2)] = left->m[XY(5, 2)];
	parent->m[XY(1, 3)] = left->m[XY(5, 3)];

	parent->m[XY(2, 0)] = left->m[XY(2, 4)];
	parent->m[XY(2, 1)] = left->m[XY(2, 5)];
	parent->m[XY(2, 2)] = left->m[XY(2, 2)];
	parent->m[XY(2, 3)] = left->m[XY(2, 3)];
	parent->m[XY(3, 0)] = left->m[XY(3, 4)];
	parent->m[XY(3, 1)] = left->m[XY(3, 5)];
	parent->m[XY(3, 2)] = left->m[XY(3, 2)];
	parent->m[XY(3, 3)] = left->m[XY(3, 3)];
}

__global__ void mergeRightChild(Node* nodes, int startIdx, int nodesCount)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= nodesCount)
		return;
	int nodeIdx = startIdx + idx;
	Node* parent = &nodes[nodeIdx];
	Node* right = &nodes[RIGHT(nodeIdx)];
	parent->m[XY(0, 0)] += right->m[XY(2, 2)];
	parent->m[XY(0, 1)] += right->m[XY(2, 3)];
	parent->m[XY(1, 0)] += right->m[XY(3, 2)];
	parent->m[XY(1, 1)] += right->m[XY(3, 3)];

	parent->m[XY(0, 4)] = right->m[XY(2, 4)];
	parent->m[XY(0, 5)] = right->m[XY(2, 5)];
	parent->m[XY(1, 4)] = right->m[XY(3, 4)];
	parent->m[XY(1, 5)] = right->m[XY(3, 5)];

	parent->m[XY(4, 0)] = right->m[XY(4, 2)];
	parent->m[XY(4, 1)] = right->m[XY(4, 3)];
	parent->m[XY(4, 4)] = right->m[XY(4, 4)];
	parent->m[XY(4, 5)] = right->m[XY(4, 5)];
	parent->m[XY(5, 0)] = right->m[XY(5, 2)];
	parent->m[XY(5, 1)] = right->m[XY(5, 3)];
	parent->m[XY(5, 4)] = right->m[XY(5, 4)];
	parent->m[XY(5, 5)] = right->m[XY(5, 5)];
}

__global__ void divideLeft(Node* nodes, number* leftSide)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= dProps.bottomNodes)
		return;
	int nodeIdx = (idx < dProps.lastLevelNodes) * (dProps.lastLevelStartIdx + idx) + (idx >= dProps.lastLevelNodes) * (dProps.beforeLastLevelStartIdx + idx);
	//printf("%d %d\n", idx, nodeIdx);
	Node node = nodes[nodeIdx];
	idx *= 5 * 3;

	node.m[XY(1, 1)] = leftSide[idx + 12];
	node.m[XY(1, 2)] = leftSide[idx + 10];
	node.m[XY(1, 3)] = leftSide[idx + 11];
	node.m[XY(1, 4)] = leftSide[idx + 13];
	node.m[XY(1, 5)] = leftSide[idx + 14];

	node.m[XY(2, 1)] = leftSide[idx + 4];
	node.m[XY(2, 2)] = leftSide[idx + 2] / 2.0;
	node.m[XY(2, 3)] = leftSide[idx + 3] / 2.0;

	node.m[XY(3, 1)] = leftSide[idx + 8];
	node.m[XY(3, 2)] = leftSide[idx + 6] / 2.0;
	node.m[XY(3, 3)] = leftSide[idx + 7] / 2.0;
	node.m[XY(3, 4)] = leftSide[idx + 9];

	node.m[XY(4, 1)] = leftSide[idx + 16];

	node.m[XY(4, 3)] = leftSide[idx + 15];
	node.m[XY(4, 4)] = leftSide[idx + 17] / 2.0;
	node.m[XY(4, 5)] = leftSide[idx + 18] / 2.0;

	node.m[XY(5, 1)] = leftSide[idx + 20];


	node.m[XY(5, 4)] = leftSide[idx + 21] / 2.0;
	node.m[XY(5, 5)] = leftSide[idx + 22] / 2.0;


	nodes[nodeIdx] = node;
	//	printNode(node);
}

__global__ void divideFirstAndLast(Node* nodes, number* leftSide)
{
	int nodeIdx = dProps.lastLevelStartIdx;
	nodes[nodeIdx].m[XY(2, 2)] = leftSide[2];
	nodes[nodeIdx].m[XY(2, 3)] = leftSide[3];

	nodes[nodeIdx].m[XY(3, 2)] = leftSide[6];
	nodes[nodeIdx].m[XY(3, 3)] = leftSide[7];

	nodeIdx = (dProps.beforeLastLevelNodes == 0) * (dProps.heapNodes - 1) + (dProps.beforeLastLevelNodes != 0) * (dProps.heapNodes - dProps.lastLevelNodes - 1);
	nodes[nodeIdx].m[XY(4, 4)] = leftSide[dProps.leftSize - 25 + 17];
	nodes[nodeIdx].m[XY(4, 5)] = leftSide[dProps.leftSize - 25 + 18];

	nodes[nodeIdx].m[XY(5, 4)] = leftSide[dProps.leftSize - 25 + 21];
	nodes[nodeIdx].m[XY(5, 5)] = leftSide[dProps.leftSize - 25 + 22];
	//	printf("|%d %d|\n", dProps.lastLevelStartIdx, nodeIdx);
}

inline __device__ void divideRightNode(Node* nodes, number* rightSide, int ord, int nodeIdx, int idx, int rightCount)
{
	Node* node = &nodes[nodeIdx];
	rightSide += ord * 3 * rightCount;
	idx %= rightCount;
	node->x[1][idx] = (rightSide + rightCount * 2)[idx]; //n+2  //swapped first and third row, and then second and third
	node->x[2][idx] = rightSide[idx] / (1 * (ord == 0) + 2 * (ord > 0));//n TODO extract expressions to different function
	node->x[3][idx] = (rightSide + rightCount)[idx] / (1 * (ord == 0) + 2 * (ord > 0)); //n+1
	node->x[4][idx] = (rightSide + rightCount * 3)[idx] / (1 * (ord == dProps.bottomNodes - 1) + 2 * (ord < dProps.bottomNodes - 1)); //n+3
	node->x[5][idx] = (rightSide + rightCount * 4)[idx] / (1 * (ord == dProps.bottomNodes - 1) + 2 * (ord < dProps.bottomNodes - 1)); //n+4	
}

__global__ void divideRight(Node* nodes, number* rightSide)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= dProps.bottomNodes * dProps.rightCount)
		return;
	int ord = idx / dProps.rightCount;
	int nodeIdx = (ord < dProps.lastLevelNodes) * (dProps.lastLevelStartIdx + ord) + (ord >= dProps.lastLevelNodes) * (dProps.beforeLastLevelStartIdx + ord);
	divideRightNode(nodes, rightSide, ord, nodeIdx, idx, dProps.rightCount);
}

inline __device__ void assignRightNodeMem(Node* nodes, number* rightSideMem, int nodeIdx, Properties props)
{
	Node* node = &nodes[nodeIdx];
	int start = nodeIdx * props.rightCount * 6;
	node->x[0] = rightSideMem + start;
	node->x[1] = rightSideMem + start + props.rightCount;
	node->x[2] = rightSideMem + start + props.rightCount * 2;
	node->x[3] = rightSideMem + start + props.rightCount * 3;
	node->x[4] = rightSideMem + start + props.rightCount * 4;
	node->x[5] = rightSideMem + start + props.rightCount * 5;
}

__global__ void assignRightSideMem(Node* nodes, number* rightSideMem)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= dProps.heapNodes)
		return;
	assignRightNodeMem(nodes, rightSideMem, idx, dProps);
}

inline __device__ void mergeRightSideNode(Node* nodes, int idx, int nodeIdx)
{
	Node* parent = &nodes[nodeIdx];
	Node* left = &nodes[LEFT(nodeIdx)];
	Node* right = &nodes[RIGHT(nodeIdx)];
	idx %= dProps.rightCount;
	parent->x[0][idx] = left->x[4][idx] + right->x[2][idx];
	parent->x[1][idx] = left->x[5][idx] + right->x[3][idx];
	parent->x[2][idx] = left->x[2][idx];
	parent->x[3][idx] = left->x[3][idx];
	parent->x[4][idx] = right->x[4][idx];
	parent->x[5][idx] = right->x[5][idx];
}

__global__ void mergeRightSideLayer(Node* nodes, int startNode, int rightSidesCount)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= rightSidesCount)
		return;
	int nodeIdx = startNode + idx / dProps.rightCount;
	mergeRightSideNode(nodes, idx, nodeIdx);
}

inline __device__ __host__ void assignParentToChildren(Node* nodes, int nodeIdx)
{
	Node* parent = &nodes[nodeIdx];
	Node* left = &nodes[LEFT(nodeIdx)];
	Node* right = &nodes[RIGHT(nodeIdx)];
	left->x[2] = parent->x[2]; //it's enough to assign pointers because it won't be modified 
	left->x[3] = parent->x[3];
	left->x[4] = parent->x[0];
	left->x[5] = parent->x[1];

	right->x[2] = parent->x[0];
	right->x[3] = parent->x[1];
	right->x[4] = parent->x[4];
	right->x[5] = parent->x[5];
}

__global__ void assignParentToChildrenLayer(Node* nodes, int startNode, int nodesCount)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= nodesCount)
		return;
	idx += startNode;
	assignParentToChildren(nodes, idx);
}

void distributeInputAmongNodes(Node* dNodes, number* dLeftSide, number* dRightSideMem, number* dRightSide, Properties props)
{
	divideLeft << <BLOCKS(props.bottomNodes), THREADS >> >(dNodes, dLeftSide);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	divideFirstAndLast << <1, 1 >> >(dNodes, dLeftSide);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	assignRightSideMem << <BLOCKS(props.heapNodes), THREADS >> >(dNodes, dRightSideMem);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	divideRight<<<BLOCKS(props.bottomNodes*props.rightCount),THREADS>>>(dNodes, dRightSide);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
}

void eliminateFirstRow(Node* dNodes, Properties props) //5x5 matrices
{
	forwardEliminationLeft << <BLOCKS(props.lastLevelNodes), THREADS >> >(dNodes, props.lastLevelStartIdx, props.lastLevelNodes, 1, 2);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	forwardEliminationRight << <BLOCKS(props.lastLevelNodes*(props.rightCount / COLUMNS_PER_THREAD)), THREADS >> >(dNodes, props.lastLevelStartIdx, props.lastLevelNodes, 1, 2);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	if (props.beforeLastLevelNodes > 0)
	{
		forwardEliminationLeft << <BLOCKS(props.beforeLastLevelNodes), THREADS >> >(dNodes, props.remainingNodes, props.beforeLastLevelNodes, 1, 2);
		ERRCHECK(hipGetLastError());
		ERRCHECK(hipDeviceSynchronize());
		forwardEliminationRight << <BLOCKS(props.beforeLastLevelNodes*(props.rightCount / COLUMNS_PER_THREAD)), THREADS >> >(dNodes, props.remainingNodes, props.beforeLastLevelNodes, 1, 2);
		ERRCHECK(hipGetLastError());
		ERRCHECK(hipDeviceSynchronize());
	}
}

void eliminateRoot(Node* dNodes, Properties props)
{
	//	mergeRightSideLayer << <1, props.rightCount>> >(dNodes, 0, props.rightCount); less than 1024
	mergeRightSideLayer << <BLOCKS(props.rightCount), THREADS>> >(dNodes, 0, props.rightCount);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	mergeLeftChild << <1,1>> >(dNodes, 0, 1);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	mergeRightChild << <1,1>> >(dNodes, 0, 1);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	forwardEliminationLeft << <1,1>> >(dNodes, 0, 1, 0, 6);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	forwardEliminationRight << <BLOCKS((props.rightCount / COLUMNS_PER_THREAD)), THREADS >> >(dNodes, 0, 1, 0, 6);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	backwardSubstitutionRight<<<BLOCKS((props.rightCount / COLUMNS_PER_THREAD)), THREADS >>>(dNodes, 0, 1, 0, 4);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	assignParentToChildrenLayer<<<1,1>>>(dNodes, 0, 1);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
}


void run(Node* dNodes, number* dLeftSide, Properties props, number* dRightSide, number* dRightSideMem)
{
	distributeInputAmongNodes(dNodes, dLeftSide, dRightSideMem, dRightSide, props);
	eliminateFirstRow(dNodes, props);
	int nodesCount = props.beforeLastLevelNotBottomNodes;

	for (int start = PARENT(props.lastLevelStartIdx); start > 0; nodesCount = (start + 1) / 2 , start = PARENT(start))//order matters
	{
		mergeRightSideLayer<<<BLOCKS(nodesCount*props.rightCount), THREADS>>>(dNodes, start, nodesCount * props.rightCount);
		ERRCHECK(hipGetLastError());
		ERRCHECK(hipDeviceSynchronize());
		mergeLeftChild << <BLOCKS(nodesCount), THREADS >> >(dNodes, start, nodesCount);
		ERRCHECK(hipGetLastError());
		ERRCHECK(hipDeviceSynchronize());
		mergeRightChild << <BLOCKS(nodesCount), THREADS >> >(dNodes, start, nodesCount);
		ERRCHECK(hipGetLastError());
		ERRCHECK(hipDeviceSynchronize());
		forwardEliminationLeft << <BLOCKS(nodesCount), THREADS >> >(dNodes, start, nodesCount, 0, 2);
		ERRCHECK(hipGetLastError());
		ERRCHECK(hipDeviceSynchronize());
		forwardEliminationRight << <BLOCKS(nodesCount*(props.rightCount / COLUMNS_PER_THREAD)), THREADS >> >(dNodes, start, nodesCount, 0, 2);
		ERRCHECK(hipGetLastError());
		ERRCHECK(hipDeviceSynchronize());
	}
	eliminateRoot(dNodes, props);
	nodesCount = props.heapNodes == 5 ? 1 : 2; //for smallest size tree is 1, otherwise 2
	for (int start = 1; start < PARENT(props.lastLevelStartIdx); start = LEFT(start) , nodesCount *= 2)
	{
		backwardSubstitutionRight<<<BLOCKS(nodesCount*(props.rightCount / COLUMNS_PER_THREAD)), THREADS >>>(dNodes, start, nodesCount, 0, 1);
		ERRCHECK(hipGetLastError());
		ERRCHECK(hipDeviceSynchronize());
		assignParentToChildrenLayer << <BLOCKS(nodesCount),THREADS>> >(dNodes, start, nodesCount);
		ERRCHECK(hipGetLastError());
		ERRCHECK(hipDeviceSynchronize());
	}

	backwardSubstitutionRight <<<BLOCKS(props.beforeLastLevelNotBottomNodes*(props.rightCount / COLUMNS_PER_THREAD)), THREADS >> >(dNodes, PARENT(props.lastLevelStartIdx), props.beforeLastLevelNotBottomNodes, 0, 1);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	if (props.beforeLastLevelNodes > 0)
	{
		backwardSubstitutionRight << <BLOCKS(props.beforeLastLevelNodes*(props.rightCount / COLUMNS_PER_THREAD)), THREADS >> >(dNodes, props.remainingNodes, props.beforeLastLevelNodes, 1, 1);
		ERRCHECK(hipGetLastError());
		ERRCHECK(hipDeviceSynchronize());
	}
	assignParentToChildrenLayer << <BLOCKS(props.beforeLastLevelNotBottomNodes), THREADS >> >(dNodes, PARENT(props.lastLevelStartIdx), props.beforeLastLevelNotBottomNodes);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	backwardSubstitutionRight << <BLOCKS(props.lastLevelNodes*(props.rightCount / COLUMNS_PER_THREAD)), THREADS >> >(dNodes, props.lastLevelStartIdx, props.lastLevelNodes, 1, 1);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
}

const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;
const int NUM_REPS = 100;


__global__ void transpose32(number* out, const number* in, unsigned dim0, unsigned dim1)
{
	__shared__ number shrdMem[TILE_DIM][TILE_DIM + 1];

	unsigned lx = threadIdx.x;
	unsigned ly = threadIdx.y;

	unsigned gx = lx + blockDim.x * blockIdx.x;
	unsigned gy = ly + TILE_DIM * blockIdx.y;

#pragma unroll
	for (unsigned repeat = 0; repeat < TILE_DIM; repeat += blockDim.y)
	{
		unsigned gy_ = gy + repeat;
		if (gx < dim0 && gy_ < dim1)
			shrdMem[ly + repeat][lx] = in[gy_ * dim0 + gx];
	}
	__syncthreads();

	gx = lx + blockDim.x * blockIdx.y;
	gy = ly + TILE_DIM * blockIdx.x;

#pragma unroll
	for (unsigned repeat = 0; repeat < TILE_DIM; repeat += blockDim.y)
	{
		unsigned gy_ = gy + repeat;
		if (gx < dim1 && gy_ < dim0)
			out[gy_ * dim0 + gx] = shrdMem[lx][ly + repeat];
	}
}

__global__ void copyRightSideBack(Node* bottomNodes, number* rightSide)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= dProps.bottomNodes * dProps.rightCount)
		return;
	int ordIdx = idx / dProps.rightCount;
	int nodeIdx = (ordIdx < dProps.lastLevelNodes) * (ordIdx + dProps.beforeLastLevelNodes) + (ordIdx >= dProps.lastLevelNodes) * (ordIdx - dProps.lastLevelNodes);
	//	printf("nodeIdx %d, %d\n", nodeIdx, idx);
	Node node = bottomNodes[nodeIdx];
	int i = idx % dProps.rightCount;
	rightSide[ordIdx * dProps.rightCount * 3 + i] = node.x[2][i];
	rightSide[ordIdx * dProps.rightCount * 3 + dProps.rightCount + i] = node.x[3][i];
	rightSide[ordIdx * dProps.rightCount * 3 + dProps.rightCount * 2 + i] = node.x[1][i];
}

// size-2 must be divisible by 3 without remainder
void runComputing(const int size, int iters, char* bitmapPath = nullptr)
{
	Properties props = getProperities(size, size);
	ERRCHECK(hipMemcpyToSymbol(HIP_SYMBOL(dProps), &props, sizeof(Properties)));
	number* leftSide = nullptr;
	number* rightSide = nullptr;
	number* dRightSideCopy = nullptr;
	number* dRightSideMem = nullptr;
	number* rightSideMem = new number[dProps.rightSizeMem];
	number* leftSideCopy = new number[dProps.leftSize];
	memcpy(leftSideCopy, leftSide, dProps.leftSize);
	Node* nodes = new Node[props.heapNodes];
	memset(nodes, 0, props.heapNodes * sizeof(Node));
	Node* dNodes = nullptr;
	number* dLeftSide = nullptr;
	number* dRightSide = nullptr;
	if (bitmapPath == nullptr)
		generateTestEquation(size, props.rightCount, &leftSide, &rightSide);
	ERRCHECK(hipMalloc(&dNodes, sizeof(Node)* props.heapNodes));
	ERRCHECK(hipMalloc(&dLeftSide, sizeof(number)*props.leftSize));
	ERRCHECK(hipMalloc(&dRightSide, sizeof(number)*props.rightSize));
	ERRCHECK(hipMalloc(&dRightSideCopy, sizeof(number)*props.rightSize));
	ERRCHECK(hipMalloc(&dRightSideMem, sizeof(number)*props.rightSizeMem));
	clock_t start, end;
	start = clock();
	for (int i = 0; i < iters; i++)
	{
		if (bitmapPath != nullptr)
		{
			BSpline2d bSplines;
			rightSide = generateBitmapRightSide(bitmapPath, size - 2, &bSplines);
//			rightSide = cutSquare(rightSide, size, props.rightCount);
//			printf("begpoint\n");
//			for (int i = 0; i < props.leftCount; i++)
//			{
//				for (int j = 0; j < props.rightCount; j++)
//				{
//					printf("%.2f ", rightSide[i * props.rightCount + j]);
//				}
//				printf("\n");
//			}
//			printf("begpointend\n");
			leftSide = generateBitmapLeftSide(bSplines, size);
			printLeftAndRight(leftSide, rightSide, size,props.rightCount);
		}

		ERRCHECK(hipMemset(dNodes, 0, sizeof(Node)*props.heapNodes));
		ERRCHECK(hipMemcpy(dLeftSide, leftSide, sizeof(number)*props.leftSize, hipMemcpyHostToDevice));
		ERRCHECK(hipMemcpy(dRightSide, rightSide, sizeof(number)*props.rightSize, hipMemcpyHostToDevice));
		//		showMemoryConsumption();
		run(dNodes, dLeftSide, props, dRightSide, dRightSideMem);
		//copy last two right side rows
		//PUT RESULT INTO ONE ARRAY
		copyRightSideBack << <BLOCKS(props.bottomNodes*props.rightCount), THREADS >> >(dNodes + props.remainingNodes, dRightSideCopy);
		ERRCHECK(hipGetLastError());
		ERRCHECK(hipDeviceSynchronize());//copy last two right side rows
		ERRCHECK(hipMemcpy(dRightSideCopy + props.rightSize - 2 * props.rightCount, dRightSideMem + 4 * props.rightCount, sizeof(number) * 2 * props.rightCount, hipMemcpyDeviceToDevice));
		ERRCHECK(hipGetLastError());
		ERRCHECK(hipDeviceSynchronize());
		//		COPY RIGHT SIDE BACK END
//		printf("midpoint\n");
//		ERRCHECK(hipMemcpy(rightSide, dRightSideCopy, sizeof(number)*props.rightSize, hipMemcpyDeviceToHost));
//		for (int i = 0; i < props.leftCount; i++)
//		{
//			for (int j = 0; j < props.rightCount; j++)
//			{
//				printf("%.4f ", rightSide[i * props.rightCount + j]);
//			}
//			printf("\n");
//		}
//		printf("midpointend\n");
		transpose32 << <dim3((props.leftCount + TILE_DIM) / TILE_DIM, (props.rightCount + TILE_DIM) / TILE_DIM), dim3(TILE_DIM, BLOCK_ROWS) >> >(dRightSide, dRightSideCopy, props.leftCount, props.rightCount);
		ERRCHECK(hipGetLastError());
		ERRCHECK(hipDeviceSynchronize());
//		printf("transpoint\n");
		ERRCHECK(hipMemcpy(rightSide, dRightSide, sizeof(number)*props.rightSize, hipMemcpyDeviceToHost));
//		for (int i = 0; i < props.leftCount; i++)
//		{
//			for (int j = 0; j < props.rightCount; j++)
//			{
//				printf("%.2f ", rightSide[i * props.rightCount + j]);
//			}
//			printf("\n");
//		}
//		printf("transpointend\n");
		//RUN FOR TRANSPOSED MATRIX
//		ERRCHECK(hipMemcpy(leftSide, dLeftSide, sizeof(number)*props.leftSize, hipMemcpyDeviceToHost));
		printLeftAndRight(leftSide, rightSide, size, props.rightCount);
		run(dNodes, dLeftSide, props, dRightSide, dRightSideMem);
		//PUT RESULT INTO ONE ARRAY
		copyRightSideBack << <BLOCKS(props.bottomNodes*props.rightCount), THREADS >> >(dNodes + props.remainingNodes, dRightSideCopy);
		ERRCHECK(hipGetLastError());
		ERRCHECK(hipDeviceSynchronize());
		ERRCHECK(hipMemcpy(dRightSideCopy + props.rightSize - 2 * props.rightCount, dRightSideMem + 4 * props.rightCount, sizeof(number) * 2 * props.rightCount, hipMemcpyDeviceToDevice));//copy last two right side rows
		ERRCHECK(hipGetLastError());
		ERRCHECK(hipDeviceSynchronize());
		printf("endpoint\n");
		ERRCHECK(hipMemcpy(rightSide, dRightSideCopy, sizeof(number)*props.rightSize, hipMemcpyDeviceToHost));
		for (int i = 0; i < props.leftCount; i++)
		{
			for (int j = 0; j < props.rightCount; j++)
			{
				printf("%.4f ", rightSide[i * props.rightCount + j]);
			}
			printf("\n");
		}
		printf("endpointend\n");
		if (bitmapPath != nullptr)
		{
			delete[] rightSide;
			delete[] leftSide;
		}
	}
	end = clock();
	printf("time %f\n", ((number)(end - start) / CLOCKS_PER_SEC) / iters);
	//	ERRCHECK(hipMemcpy(rightSide, dRightSide, sizeof(number)*props.rightSize, hipMemcpyDeviceToHost));
	//	for (int i = 0; i < props.leftCount; i++)
	//	{
	//		for (int j = 0; j < props.rightCount; j++)
	//		{
	//			printf("%.2f ", rightSide[i * props.rightCount + j]);
	//		}
	//		printf("\n");
	//	}
	//	printf("\n");
	number* solution = new number[size * size];
	hipMemcpy(solution, dRightSideCopy, sizeof(number) * props.rightSize, hipMemcpyDeviceToHost);
	getBitmapApprox(solution, size - 2, size-2);
	//run(dNodes, dLeftSide, props, dRightSide, dRightSideMem);

	//	ERRCHECK(hipMemcpy(nodes, dNodes, sizeof(Node) * props.heapNodes, hipMemcpyDeviceToHost));
	//	ERRCHECK(hipMemcpy(rightSideMem, dRightSideMem, sizeof(number)*props.rightSizeMem, hipMemcpyDeviceToHost));
	if (bitmapPath == nullptr)
	{
		delete[] leftSide;
		delete[] rightSide;
	}
	delete[] rightSideMem;
	delete[] nodes;
	ERRCHECK(hipFree(dRightSide));
	ERRCHECK(hipFree(dRightSideCopy));
	ERRCHECK(hipFree(dRightSideMem));
	ERRCHECK(hipFree(dNodes));
	ERRCHECK(hipFree(dLeftSide));
}

int main()
{
	//	generate2DSplineIntegrals(9, 3);
	//	readBmp("C:/Users/quirell/Pictures/Untitled.bmp");
	//	generateBitmapRightSide("C:/Users/quirell/Desktop/magisterka/bitmapy/e253ppe15white.bmp",253);
	//	measureGenBitmap("C:/Users/quirell/Desktop/magisterka/bitmapy/e510ppe20.bmp", 510, 1);
	runComputing(14, 1, "C:/Users/quirell/Desktop/magisterka/bitmapy/4colors5px12.bmp");
//	runComputing(11, 1);
	//	runComputing(255, 10, "C:/Users/quirell/Desktop/magisterka/bitmapy/e253ppe40.bmp");
	//	runComputing(255, 1, "C:/Users/quirell/Desktop/magisterka/bitmapy/e510ppe40.bmp");
	//		runComputing(65, 100);
	//		runComputing(128, 100);
	//		runComputing(255, 100);
	//		runComputing(512, 100);
	//		runComputing(1022, 100);
	//		runComputing(2048, 100);
	//		runComputing(4097, 100);
	//		runComputing(6146, 30);
	//		runComputing(8192, 30);
	//		runComputing(65, 1);
	//		runComputing(128, 1);
	//		runComputing(255, 1);
	//		runComputing(512, 1);
	//		runComputing(1022, 1);
	//		runComputing(2048, 1);
	//		runComputing(4097, 1);
	//		runComputing(6146, 1);
	//		runComputing(8192, 1);
	//		runComputing(65, 30, "C:/Users/quirell/Desktop/magisterka/bitmapy/e63ppe15.bmp");
	//		runComputing(128, 30, "C:/Users/quirell/Desktop/magisterka/bitmapy/e126ppe15.bmp");
	//		runComputing(255, 1, "C:/Users/quirell/Desktop/magisterka/bitmapy/e253ppe15white.bmp");
	//		runComputing(512, 15, "C:/Users/quirell/Desktop/magisterka/bitmapy/e510ppe15.bmp");
	//		runComputing(1022,1, "C:/Users/quirell/Desktop/magisterka/bitmapy/e1020ppe15.bmp");
	//			number * left;
	//			number * right;
	//			generateTestEquation(14, 1, &left, &right);
	//			testRun(44);
	//	testMultipleRun(1,1022);
	//	getch();
	//	testDistributeInputAmongNodes();
	//		getch();
	//			ERRCHECK(hipSetDevice(0));
	//			testGaussianElimination();
	//	number * x, * y;
	//	int s = 8129;
	//	generateTestEquation(s,s, &x,&y);
	getch();
	return 0;
}
