#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <conio.h>
#include <ctime>
#include <new>
#include <cmath>
#include <cstring>
#define ERRCHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true, bool wait = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (wait) getch();
		if (abort) exit(code);
	}
}

#define MSIZE 36
#define INPUT_SIZE(N) N*5 - 3*2
#define PARENT(i) (i-1)/2
#define LEFT(i) 2*i + 1
#define RIGHT(i) 2*i + 2
#define BOTTOM_HEAP_NODES_COUNT(N) (N-2)/3 //size of input must be 2+3n,n>1
#define HEAP_SIZE(N) 2*BOTTOM_HEAP_NODES_COUNT(N)-1
#define FIRST_LEVEL_SIZE 19
#define ROW_LENGTH 5
#define FIRST_LVL_MAT_SIZE 5
#define XY(x,y) x*6+y
#define THREADS 512
#define BLOCKS(N) (N+THREADS)/THREADS
#define COLUMNS_PER_THREAD 1

const struct Properties
{
	int leftCount;
	int leftSize;
	int heapNodes;
	int bottomNodes;
	int remainingNodes;
	int lastLevelNodes;
	int beforeLastLevelNodes;
	int lastLevelStartIdx;
	int beforeLastLevelStartIdx;
	int rightCount;
	int beforeLastLevelNotBottomNodes;
	int rightSize;
};

Properties getProperities(int leftCount, int rightCount)
{
	Properties p;
	p.leftCount = leftCount;
	p.leftSize = leftCount * 5;
	p.heapNodes = HEAP_SIZE(leftCount);
	p.bottomNodes = BOTTOM_HEAP_NODES_COUNT(leftCount);
	p.remainingNodes = p.heapNodes - p.bottomNodes;
	p.beforeLastLevelStartIdx = (int)pow(2, (int)log2(p.remainingNodes)) - 1;
	p.beforeLastLevelNotBottomNodes = p.remainingNodes - p.beforeLastLevelStartIdx;
	p.beforeLastLevelNodes = pow(2, (int)log2(p.bottomNodes - 1)) - p.beforeLastLevelNotBottomNodes;// -1 is in case bottomNodes is power of two, then beforeLastLevelNodes should obviously be 0
	p.lastLevelNodes = p.bottomNodes - p.beforeLastLevelNodes;
	p.lastLevelStartIdx = p.heapNodes - p.lastLevelNodes;
	p.beforeLastLevelStartIdx = p.remainingNodes - p.lastLevelNodes; //account for idx value, undefined when beforeLastLevelNodes is = 0
	p.rightCount = rightCount;
	p.rightSize = rightCount * leftCount;
	return p;
}


__constant__ Properties dProps;

struct Node
{
	float m[MSIZE];
	float* x[6];
};

void printAllNodes(Node* nodes, int nodesStart, Properties props);
__device__ __host__ void printNode(Node node);


void fillRightSide(float value, int row, float* rightSide, int rightCount)
{
	for (int i = 0; i < rightCount; i++)
	{
		rightSide[row * rightCount + i] = value;
	}
}

void generateTestEquation(int leftCount, int rightCount, float** leftSidePtr, float** rightSidePtr)
{
	float* leftSide = (float*)malloc(sizeof(float) * leftCount * 5);
	float* rightSide = (float*)malloc(sizeof(float) * rightCount * leftCount);
	for (int i = 0; i < leftCount * 5; i++)
		leftSide[i] = 6;// i / 5 + 1;
	leftSide[0] = 0;
	leftSide[1] = 0;
	leftSide[5] = 0;
	leftSide[leftCount * 5 - 6] = 0;
	leftSide[leftCount * 5 - 2] = 0;
	leftSide[leftCount * 5 - 1] = 0;

	for (int i = 2; i < leftCount - 2; i++)
	{
		int rightSideVal = 0;
		for (int j = 0; j < 5; j++)
		{
			int solution = (i - 1) + j; //solution is x(0)=1,x(1)=2,x(n-1)=n
			rightSideVal += 6 * solution;
		}
		fillRightSide(rightSideVal, i, rightSide, rightCount);
	}
	fillRightSide(1 * 6 + 2 * 6 + 3 * 6, 0, rightSide, rightCount);
	fillRightSide(1 * 6 + 2 * 6 + 3 * 6 + 4 * 6, 1, rightSide, rightCount);
	fillRightSide((leftCount - 3) * 6 + (leftCount - 2) * 6 + (leftCount - 1) * 6 + leftCount * 6, leftCount - 2, rightSide, rightCount);
	fillRightSide((leftCount - 2) * 6 + (leftCount - 1) * 6 + leftCount * 6, leftCount - 1, rightSide, rightCount);
	*leftSidePtr = leftSide;
	*rightSidePtr = rightSide;
	//		for (int i = 0; i < leftCount; i++)
	//		{
	//			printf("%d:", i + 1);
	//			for (int j = 0; j < 5; j++)
	//			{
	//				printf("%.0f ", leftSide[i * 5 + j]);
	//			}
	//			printf(" |  ");
	//			for (int j = 0; j < rightCount; j++)
	//			{
	//				printf("%.0f ", rightSide[i * rightCount + j]);
	//			}
	//			printf("\n");
	//		}
	//		getch();
}

__global__ void backwardSubstitutionRight(Node* nodes, int startIdx, int nodesCount, int end, int elim)
{
	int idx = (blockIdx.x * blockDim.x + threadIdx.x) / (dProps.rightCount / COLUMNS_PER_THREAD);
	if (idx >= nodesCount)
		return;
	int colStart = ((blockIdx.x * blockDim.x + threadIdx.x) % (dProps.rightCount / COLUMNS_PER_THREAD)) * COLUMNS_PER_THREAD;
	//	printf("%d %d\n", idx, colStart);
	int nodeIdx = startIdx + idx;
	float* m = nodes[nodeIdx].m;
	float** x = nodes[nodeIdx].x;
	for (int rcol = colStart; rcol < colStart + COLUMNS_PER_THREAD; rcol++)
	{
		for (int row = elim; row >= end; row--)//max elim == 4,5th is already done after elimination
		{
			for (int col = row + 1; col < 6; col++)
			{
				x[row][rcol] -= m[XY(row, col)] * x[col][rcol];
			}
		}
	}
}

__global__ void forwardEliminationLeft(Node* nodes, int startIdx, int nodesCount, int start, int elim)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= nodesCount)
		return;
	int nodeIdx = startIdx + idx;
	float* m = nodes[nodeIdx].m;
	for (int row = start; row < elim; row++)
	{
		for (int col = row + 1; col < 6; col++) //from element after diagonal
		{
			m[XY(row, col)] /= m[XY(row, row)];
		}
		for (int rowBelow = row + 1; rowBelow < 6; rowBelow++)
		{
			for (int col = row + 1; col < 6; col++)
			{
				m[XY(rowBelow, col)] -= m[XY(rowBelow, row)] * m[XY(row, col)];
			}
		}
	}
}

__global__ void forwardEliminationRight(Node* nodes, int startIdx, int nodesCount, int rowStart, int elim)
{
	int idx = (blockIdx.x * blockDim.x + threadIdx.x) / (dProps.rightCount / COLUMNS_PER_THREAD);
	if (idx >= nodesCount)
		return;
	int colStart = ((blockIdx.x * blockDim.x + threadIdx.x) % (dProps.rightCount / COLUMNS_PER_THREAD)) * COLUMNS_PER_THREAD;
	int nodeIdx = startIdx + idx;
	float* m = nodes[nodeIdx].m;
	float** x = nodes[nodeIdx].x;
	for (int row = rowStart; row < elim; row++)
	{
		for (int col = colStart; col < colStart + COLUMNS_PER_THREAD; col++)
		{
			x[row][col] /= m[XY(row, row)];
		}
		for (int rowBelow = row + 1; rowBelow < 6; rowBelow++)
		{
			for (int col = colStart; col < colStart + COLUMNS_PER_THREAD; col++)
			{
				x[rowBelow][col] -= m[XY(rowBelow, row)] * x[row][col];
			}
		}
	}
}

__global__ void assignTestRightSize(Node* node, float* x)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= 1)
		return;
	node->x[0] = x;
	node->x[1] = x + 4;
	node->x[2] = x + 4 * 2;
	node->x[3] = x + 4 * 3;
	node->x[4] = x + 4 * 4;
	node->x[5] = x + 4 * 5;
}

void testGaussianElimination()
{
	Properties props = getProperities(1, 4);
	ERRCHECK(hipMemcpyToSymbol(HIP_SYMBOL(dProps), &props, sizeof(Properties)));
	Node node;
	float m[] = {
		1, 1, -2, 1, 3, -1,
		2, -1, 1, 2, 1, -3,
		1, 3, -3, -1, 2, 1,
		5, 2, -1, -1, 2, 1,
		-3, -1, 2, 3, 1, 3,
		4, 3, 1, -6, -3, -2
	};
	memcpy(node.m, m, sizeof(float) * MSIZE);
	Node* dNode;
	printNode(node);
	ERRCHECK(hipMalloc(&dNode, sizeof(Node)));
	ERRCHECK(hipMemcpy(dNode, &node, sizeof(Node), hipMemcpyHostToDevice));
	float x[] = {4,4,4,4,20,20,20,20,-15,-15,-15,-15,-3,-3,-3,-3,16,16,16,16,-27,-27,-27,-27};
	float* dX;
	ERRCHECK(hipMalloc(&dX, sizeof(x)));
	ERRCHECK(hipMemcpy(dX, &x, sizeof(x) , hipMemcpyHostToDevice));
	assignTestRightSize <<<1, 1 >>>(dNode, dX);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	forwardEliminationLeft << <1, 1 >> >(dNode, 0, 1, 0, 6);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	forwardEliminationRight << <1, 4 / COLUMNS_PER_THREAD >> >(dNode, 0, 1, 0, 6);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	backwardSubstitutionRight << <1, 4 / COLUMNS_PER_THREAD >> >(dNode, 0, 1, 0, 4);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	ERRCHECK(hipMemcpy(&node, dNode, sizeof(Node), hipMemcpyDeviceToHost));
	printNode(node);
	ERRCHECK(hipMemcpy(x, dX, sizeof(x), hipMemcpyDeviceToHost));
	float c[] = {1,-2,3,4,2,-1};
	for (int i = 0; i < 6; i++)
	{
		printf("%.1f ", c[i]);
	}
	printf("\n");
	for (int i = 0; i < props.rightCount; i++)
	{
		for (int j = 0; j < 6; j++)
			printf("%.1f ", x[j * props.rightCount + i]);
		printf("\n");
	}
}


__global__ void mergeLeftChild(Node* nodes, int startIdx, int nodesCount)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= nodesCount)
		return;
	int nodeIdx = startIdx + idx;
	Node* parent = &nodes[nodeIdx];

	Node* left = &nodes[LEFT(nodeIdx)];

	parent->m[XY(0, 0)] += left->m[XY(4, 4)];
	parent->m[XY(0, 1)] += left->m[XY(4, 5)];
	parent->m[XY(1, 0)] += left->m[XY(5, 4)];
	parent->m[XY(1, 1)] += left->m[XY(5, 5)];

	parent->m[XY(0, 2)] = left->m[XY(4, 2)];
	parent->m[XY(0, 3)] = left->m[XY(4, 3)];
	parent->m[XY(1, 2)] = left->m[XY(5, 2)];
	parent->m[XY(1, 3)] = left->m[XY(5, 3)];

	parent->m[XY(2, 0)] = left->m[XY(2, 4)];
	parent->m[XY(2, 1)] = left->m[XY(2, 5)];
	parent->m[XY(2, 2)] = left->m[XY(2, 2)];
	parent->m[XY(2, 3)] = left->m[XY(2, 3)];
	parent->m[XY(3, 0)] = left->m[XY(3, 4)];
	parent->m[XY(3, 1)] = left->m[XY(3, 5)];
	parent->m[XY(3, 2)] = left->m[XY(3, 2)];
	parent->m[XY(3, 3)] = left->m[XY(3, 3)];
}

__global__ void mergeRightChild(Node* nodes, int startIdx, int nodesCount)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= nodesCount)
		return;
	int nodeIdx = startIdx + idx;
	Node* parent = &nodes[nodeIdx];
	Node* right = &nodes[RIGHT(nodeIdx)];
	parent->m[XY(0, 0)] += right->m[XY(2, 2)];
	parent->m[XY(0, 1)] += right->m[XY(2, 3)];
	parent->m[XY(1, 0)] += right->m[XY(3, 2)];
	parent->m[XY(1, 1)] += right->m[XY(3, 3)];

	parent->m[XY(0, 4)] = right->m[XY(2, 4)];
	parent->m[XY(0, 5)] = right->m[XY(2, 5)];
	parent->m[XY(1, 4)] = right->m[XY(3, 4)];
	parent->m[XY(1, 5)] = right->m[XY(3, 5)];

	parent->m[XY(4, 0)] = right->m[XY(4, 2)];
	parent->m[XY(4, 1)] = right->m[XY(4, 3)];
	parent->m[XY(4, 4)] = right->m[XY(4, 4)];
	parent->m[XY(4, 5)] = right->m[XY(4, 5)];
	parent->m[XY(5, 0)] = right->m[XY(5, 2)];
	parent->m[XY(5, 1)] = right->m[XY(5, 3)];
	parent->m[XY(5, 4)] = right->m[XY(5, 4)];
	parent->m[XY(5, 5)] = right->m[XY(5, 5)];
}

__global__ void divideLeft(Node* nodes, float* leftSide)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= dProps.bottomNodes)
		return;
	int nodeIdx = (idx < dProps.lastLevelNodes) * (dProps.lastLevelStartIdx + idx) + (idx >= dProps.lastLevelNodes) * (dProps.beforeLastLevelStartIdx + idx);
	//printf("%d %d\n", idx, nodeIdx);
	Node node = nodes[nodeIdx];
	idx *= 5 * 3;

	node.m[XY(1, 1)] = leftSide[idx + 12];
	node.m[XY(1, 2)] = leftSide[idx + 10];
	node.m[XY(1, 3)] = leftSide[idx + 11];
	node.m[XY(1, 4)] = leftSide[idx + 13];
	node.m[XY(1, 5)] = leftSide[idx + 14];

	node.m[XY(2, 1)] = leftSide[idx + 4];
	node.m[XY(2, 2)] = leftSide[idx + 2] / 2;
	node.m[XY(2, 3)] = leftSide[idx + 3] / 2;

	node.m[XY(3, 1)] = leftSide[idx + 8];
	node.m[XY(3, 2)] = leftSide[idx + 6] / 2;
	node.m[XY(3, 3)] = leftSide[idx + 7] / 2;
	node.m[XY(3, 4)] = leftSide[idx + 9];

	node.m[XY(4, 1)] = leftSide[idx + 16];

	node.m[XY(4, 3)] = leftSide[idx + 15];
	node.m[XY(4, 4)] = leftSide[idx + 17] / 2;
	node.m[XY(4, 5)] = leftSide[idx + 18] / 2;

	node.m[XY(5, 1)] = leftSide[idx + 20];


	node.m[XY(5, 4)] = leftSide[idx + 21] / 2;
	node.m[XY(5, 5)] = leftSide[idx + 22] / 2;


	nodes[nodeIdx] = node;
	//	printNode(node);
}

__global__ void divideFirstAndLast(Node* nodes, float* leftSide)
{
	int nodeIdx = dProps.lastLevelStartIdx;
	nodes[dProps.remainingNodes].m[XY(2, 3)] = leftSide[2];
	nodes[dProps.remainingNodes].m[XY(2, 2)] = leftSide[3];

	nodes[dProps.remainingNodes].m[XY(3, 3)] = leftSide[6];
	nodes[dProps.remainingNodes].m[XY(3, 2)] = leftSide[7];

	nodeIdx = (dProps.beforeLastLevelNodes == 0) * (dProps.heapNodes - 1) + (dProps.beforeLastLevelNodes != 0) * (dProps.heapNodes - dProps.lastLevelNodes - 1);
	nodes[nodeIdx].m[XY(4, 4)] = leftSide[dProps.leftSize - 25 + 17];
	nodes[nodeIdx].m[XY(4, 5)] = leftSide[dProps.leftSize - 25 + 18];

	nodes[nodeIdx].m[XY(5, 4)] = leftSide[dProps.leftSize - 25 + 21];
	nodes[nodeIdx].m[XY(5, 5)] = leftSide[dProps.leftSize - 25 + 22];
	//printf("|%d %d|\n", dProps.lastLevelStartIdx, nodeIdx);
}

inline __device__ __host__ void divideRightNode(Node* nodes, float* rightSide, int nodeIdx, int idx, int rightCount)
{
	Node* node = &nodes[nodeIdx];
	idx *= rightCount * 3;
	rightSide += idx;
	node->x[0] = nullptr;
	node->x[1] = rightSide + rightCount * 2; //n+2  //swapped first and third row, and then second and third
	node->x[2] = rightSide;//n
	node->x[3] = rightSide + rightCount; //n+1
	node->x[4] = rightSide + rightCount * 3; //n+3
	node->x[5] = rightSide + rightCount * 4; //n+4	
}

__global__ void divideRight(Node* nodes, float* rightSide)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= dProps.bottomNodes)
		return;
	int nodeIdx = (idx < dProps.lastLevelNodes) * (dProps.lastLevelStartIdx + idx) + (idx >= dProps.lastLevelNodes) * (dProps.beforeLastLevelStartIdx + idx);
	divideRightNode(nodes, rightSide, nodeIdx, idx, dProps.rightCount);
}

inline __device__ __host__ void copyRightNode(Node* nodes, int idx)
{
	Node* node = &nodes[idx];
	node->x[0] = nodes[LEFT(idx)].x[4]; //do not require merging, because two children share the same memory  and not use it simultaneously
	node->x[1] = nodes[LEFT(idx)].x[5]; //
	node->x[2] = nodes[LEFT(idx)].x[2];
	node->x[3] = nodes[LEFT(idx)].x[3];
	node->x[4] = nodes[RIGHT(idx)].x[4];
	node->x[5] = nodes[RIGHT(idx)].x[5];
}

__global__ void copyRight(Node* nodes, int nodesCount)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= nodesCount)
		return;
	copyRightNode(nodes, idx);
}

void leftSideInit(float* leftSide, int size)
{
	for (int i = 0; i < size; i++)
	{
		leftSide[i] = 6;//(i+1)%26;
	}
}

void showMemoryConsumption()
{
	size_t free_byte;
	size_t total_byte;
	ERRCHECK(hipMemGetInfo(&free_byte, &total_byte));
	double free_db = (double)free_byte;
	double total_db = (double)total_byte;
	double used_db = total_db - free_db;

	printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n", used_db / 1024.0 / 1024.0, free_db / 1024.0 / 1024.0, total_db / 1024.0 / 1024.0);
}


void distributeInputAmongNodes(Node* dNodes, float* dLeftSide, float* dRightSide, Properties props)
{
	divideLeft << <BLOCKS(dProps.bottomNodes), THREADS >> >(dNodes, dLeftSide);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	divideFirstAndLast << <1, 1 >> >(dNodes, dLeftSide);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	ERRCHECK(hipFree(dLeftSide));
	divideRight<<<BLOCKS(dProps.bottomNodes),THREADS>>>(dNodes, dRightSide);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
}

void divideHostRightSide(Properties props, Node* nodes, float* rightSide)
{
	for (int i = props.remainingNodes; i < props.heapNodes; i++)
	{
		divideRightNode(nodes, rightSide, i, i - props.remainingNodes, props.rightCount);
	}
	for (int i = props.remainingNodes - 1; i >= 0; i--)
	{
		copyRightNode(nodes, i);
	}
}

void testDistributeInputAmongNodes()
{
	Properties props = getProperities(14, 1);
	ERRCHECK(hipMemcpyToSymbol(HIP_SYMBOL(dProps), &props, sizeof(Properties)));
	float* leftSide;
	float* rightSide;
	generateTestEquation(14, 2, &leftSide, &rightSide);
	Node* nodes = new Node[props.heapNodes];
	memset(nodes, 0, props.heapNodes * sizeof(Node));
	Node* dNodes = nullptr;
	float* dLeftSide = nullptr;
	float* dRightSide = nullptr;
	ERRCHECK(hipMalloc(&dNodes, sizeof(Node)* props.heapNodes));
	ERRCHECK(hipMemset(dNodes, 0, sizeof(Node)*props.heapNodes));
	ERRCHECK(hipMalloc(&dLeftSide, sizeof(float)*props.leftSize));
	ERRCHECK(hipMemcpy(dLeftSide, leftSide, sizeof(float)*props.leftSize, hipMemcpyHostToDevice));
	ERRCHECK(hipMalloc(&dRightSide, sizeof(float)*props.rightSize));
	ERRCHECK(hipMemcpy(dRightSide, rightSide, sizeof(float)*props.rightSize, hipMemcpyHostToDevice));
	distributeInputAmongNodes(dNodes, dLeftSide, dRightSide, props);
	for (int start = PARENT(props.lastLevelStartIdx), nodesCount = props.beforeLastLevelNotBottomNodes; start > 0; nodesCount = (start + 1) / 2 , start = PARENT(start))//order matters
	{
		copyRight << <BLOCKS(nodesCount), THREADS >> >(dNodes, nodesCount);
		ERRCHECK(hipGetLastError());
		ERRCHECK(hipDeviceSynchronize());
		mergeLeftChild << <BLOCKS(nodesCount), THREADS >> >(dNodes, start, nodesCount);
		ERRCHECK(hipGetLastError());
		ERRCHECK(hipDeviceSynchronize());
		mergeRightChild << <BLOCKS(nodesCount), THREADS >> >(dNodes, start, nodesCount);
		ERRCHECK(hipGetLastError());
		ERRCHECK(hipDeviceSynchronize());
	}
	copyRight << <1, THREADS >> >(dNodes, 1);
	mergeLeftChild << <1, 1 >> >(dNodes, 0, 1);
	mergeRightChild << <1, 1 >> >(dNodes, 0, 1);
	ERRCHECK(hipMemcpy(nodes, dNodes, sizeof(Node) * props.heapNodes, hipMemcpyDeviceToHost));
	ERRCHECK(hipMemcpy(rightSide,dRightSide, sizeof(float)*props.rightSize, hipMemcpyDeviceToHost));
	divideHostRightSide(props, nodes, rightSide);
	printAllNodes(nodes, 0, props);
}

void eliminateFirstRow(Node* dNodes, Properties props) //5x5 matrices
{
	forwardEliminationLeft << <BLOCKS(props.bottomNodes), THREADS >> >(dNodes, props.lastLevelStartIdx, props.bottomNodes, 1, 1);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	forwardEliminationRight << <BLOCKS(props.bottomNodes), THREADS >> >(dNodes, props.lastLevelStartIdx, props.bottomNodes, 1, 1);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	if (props.beforeLastLevelNodes > 0)
	{
		forwardEliminationLeft << <BLOCKS(props.beforeLastLevelNodes), THREADS >> >(dNodes, props.remainingNodes, props.beforeLastLevelNodes, 1, 1);
		ERRCHECK(hipGetLastError());
		ERRCHECK(hipDeviceSynchronize());
		forwardEliminationRight << <BLOCKS(props.beforeLastLevelNodes), THREADS >> >(dNodes, props.remainingNodes, props.beforeLastLevelNodes, 1, 1);
		ERRCHECK(hipGetLastError());
		ERRCHECK(hipDeviceSynchronize());
	}
}

void eliminateRoot(Node* dNodes, Properties props)
{
	copyRight << <1, THREADS >> >(dNodes, 1);
	mergeLeftChild << <1,1>> >(dNodes, 0, 1);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	mergeRightChild << <1,1>> >(dNodes, 0, 1);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	forwardEliminationLeft << <1,1>> >(dNodes, 0, 1, 0, 6);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	forwardEliminationRight << <BLOCKS(1*(props.rightCount / COLUMNS_PER_THREAD)), THREADS >> >(dNodes, 0, 1, 0, 6);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	backwardSubstitutionRight<<<BLOCKS(1*(props.rightCount / COLUMNS_PER_THREAD)), THREADS >>>(dNodes, 0, 1, 0, 6);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
}

void run(Node* dNodes, float* dLeftSide, Properties props, float* dRightSide)
{
	distributeInputAmongNodes(dNodes, dLeftSide, dRightSide, props);
	eliminateFirstRow(dNodes, props);
	int nodesCount = props.beforeLastLevelNotBottomNodes;

	for (int start = PARENT(props.lastLevelStartIdx); start > 0; nodesCount = (start + 1) / 2 , start = PARENT(start))//order matters
	{
		copyRight<<<BLOCKS(nodesCount), THREADS>>>(dNodes, nodesCount);
		ERRCHECK(hipGetLastError());
		ERRCHECK(hipDeviceSynchronize());
		mergeLeftChild << <BLOCKS(nodesCount), THREADS >> >(dNodes, start, nodesCount);
		ERRCHECK(hipGetLastError());
		ERRCHECK(hipDeviceSynchronize());
		mergeRightChild << <BLOCKS(nodesCount), THREADS >> >(dNodes, start, nodesCount);
		ERRCHECK(hipGetLastError());
		ERRCHECK(hipDeviceSynchronize());
		forwardEliminationLeft << <BLOCKS(nodesCount), THREADS >> >(dNodes, start, nodesCount, 0, 2);
		ERRCHECK(hipGetLastError());
		ERRCHECK(hipDeviceSynchronize());
		forwardEliminationRight << <BLOCKS(nodesCount*(props.rightCount / COLUMNS_PER_THREAD)), THREADS >> >(dNodes, start, nodesCount, 0, 2);
		ERRCHECK(hipGetLastError());
		ERRCHECK(hipDeviceSynchronize());
	}
	eliminateRoot(dNodes, props); 
	nodesCount = 2;
	for (int start = 1; start < PARENT(props.lastLevelStartIdx); start = LEFT(start) , nodesCount *= 2)
	{
		backwardSubstitutionRight<<<BLOCKS(nodesCount*(props.rightCount / COLUMNS_PER_THREAD)), THREADS >>>(dNodes, start, nodesCount, 0, 2);
		ERRCHECK(hipGetLastError());
		ERRCHECK(hipDeviceSynchronize());
	}
	backwardSubstitutionRight << <BLOCKS(nodesCount*(props.rightCount / COLUMNS_PER_THREAD)), THREADS >> >(dNodes, PARENT(props.lastLevelStartIdx), props.beforeLastLevelNotBottomNodes, 0, 2);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	if (props.beforeLastLevelNodes > 0)
	{
		backwardSubstitutionRight << <BLOCKS(nodesCount*(props.rightCount / COLUMNS_PER_THREAD)), THREADS >> >(dNodes, props.remainingNodes, dProps.beforeLastLevelNodes, 1, 1);
		ERRCHECK(hipGetLastError());
		ERRCHECK(hipDeviceSynchronize());
	}
	backwardSubstitutionRight << <BLOCKS(nodesCount*(props.rightCount / COLUMNS_PER_THREAD)), THREADS >> >(dNodes, props.lastLevelStartIdx, dProps.bottomNodes, 1, 1);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
}

void testRun()
{
	Properties props = getProperities(14, 1);
	ERRCHECK(hipMemcpyToSymbol(HIP_SYMBOL(dProps), &props, sizeof(Properties)));
	float* leftSide;
	float* rightSide;
	generateTestEquation(14, 1, &leftSide, &rightSide);
	Node* nodes = new Node[props.heapNodes];
	memset(nodes, 0, props.heapNodes * sizeof(Node));
	Node* dNodes = nullptr;
	float* dLeftSide = nullptr;
	float* dRightSide = nullptr;
	ERRCHECK(hipMalloc(&dNodes, sizeof(Node)* props.heapNodes));
	ERRCHECK(hipMemset(dNodes, 0, sizeof(Node)*props.heapNodes));
	ERRCHECK(hipMalloc(&dLeftSide, sizeof(float)*props.leftSize));
	ERRCHECK(hipMemcpy(dLeftSide, leftSide, sizeof(float)*props.leftSize, hipMemcpyHostToDevice));
	ERRCHECK(hipMalloc(&dRightSide, sizeof(float)*props.rightSize));
	ERRCHECK(hipMemcpy(dRightSide, rightSide, sizeof(float)*props.rightSize, hipMemcpyHostToDevice));
	run(dNodes, dLeftSide, props, dRightSide);
	divideHostRightSide(props, nodes, rightSide);
	printAllNodes(nodes, 0, props);

}
int main()
{
	testRun();
	return 0;
//	testGaussianElimination();
//	getch();
//	testDistributeInputAmongNodes();
//	getch();
//	return 0;
//	ERRCHECK(hipSetDevice(0));
//	testGaussianElimination();
//	getch();
//	return 0;
	clock_t start, end;
	//	int leftCount = (3*4+2)*10e5;
	int leftCount = 3 * 3 + 2;
	int rightCount = 1;
	const Properties props = getProperities(leftCount, rightCount);
	ERRCHECK(hipMemcpyToSymbol(HIP_SYMBOL(dProps), &props, sizeof(Properties)));

	float* leftSide = new float[props.leftSize];
	//	float * rightSide = new float[rightSize];
	Node* nodes = new Node[props.heapNodes];
	Node* dNodes = nullptr;
	float* dLeftSide = nullptr;
	leftSideInit(leftSide, props.leftSize);
	ERRCHECK(hipMalloc(&dNodes, sizeof(Node)* props.heapNodes));
	ERRCHECK(hipMemset(dNodes, 0, sizeof(Node)*props.heapNodes));
	ERRCHECK(hipMalloc(&dLeftSide, sizeof(float)*props.leftSize));
	ERRCHECK(hipMemcpy(dLeftSide, leftSide, sizeof(float)*props.leftSize, hipMemcpyHostToDevice));
	showMemoryConsumption();
	start = clock();
	divideLeft << <(props.bottomNodes + 512) / 512, 512 >> >(dNodes, dLeftSide);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	end = clock();
	printf("time %f\n", (float)(end - start) / CLOCKS_PER_SEC);
	divideFirstAndLast << <1, 1 >> >(dNodes, dLeftSide);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	mergeLeftChild << <1, 1 >> >(dNodes, PARENT(props.lastLevelStartIdx), 1);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	mergeRightChild << <1, 1 >> >(dNodes, PARENT(props.lastLevelStartIdx), 1);
	ERRCHECK(hipGetLastError());
	ERRCHECK(hipDeviceSynchronize());
	ERRCHECK(hipMemcpy(nodes, dNodes, sizeof(Node) * props.heapNodes, hipMemcpyDeviceToHost));
	printNode(nodes[0]);
	printNode(nodes[1]);
	printNode(nodes[2]);
	printNode(nodes[3]);
	printNode(nodes[4]);
	delete[] nodes;
	hipFree(dNodes);

	ERRCHECK(hipDeviceReset());

	getch();
	return 0;
}

void printAllNodes(Node* nodes, int nodesStart, Properties props)
{
	int powerOfTwo = (int)log2(nodesStart + 1) + 1;
	for (int i = nodesStart; i < props.heapNodes; i++)
	{
		//		if (i == powerOfTwo)
		//		{
		//			printf("level %d\n", powerOfTwo);
		//			powerOfTwo <<= 1;
		//		}
		Node node = nodes[i];
		for (int j = i >= props.remainingNodes ? 1 : 0; j < 6; j++)
		{
			printf("%.1f %.1f %.1f %.1f %.1f %.1f | ", node.m[XY(j, 0)], node.m[XY(j, 1)], node.m[XY(j, 2)], node.m[XY(j, 3)], node.m[XY(j, 4)], node.m[XY(j, 5)]);
			for (int k = 0; k < props.rightCount; k++)
			{
				printf("%.0f ", node.x[j][k]);
			}
			printf("\n");
		}
		printf("\n");
	}
}

__device__ __host__ void printNode(Node node)
{
	for (int i = 0; i < 6; i++)
		printf("%.1f %.1f %.1f %.1f %.1f %.1f\n", node.m[XY(i, 0)], node.m[XY(i, 1)], node.m[XY(i, 2)], node.m[XY(i, 3)], node.m[XY(i, 4)], node.m[XY(i, 5)]);
	printf("\n");
}
